#include "hip/hip_runtime.h"
#include "BlasCommon.h"
#include "cuda_sparse.h"

//global data
texture<float, 1> texXf;

//Helper functions
__device__  unsigned int compute_thread_index () {
	return ( blockIdx.x*blockDim.x*blockDim.y+
		blockIdx.y*blockDim.x*blockDim.y*gridDim.x+
		threadIdx.x+threadIdx.y*blockDim.x) ;
}

void bind_x_texf(float *x,unsigned int N)
{
	hipBindTexture(0,texXf,x,N*sizeof(float));
}

void  unbind_x_tex()
{
	hipUnbindTexture(texXf);
}

//kernel functions
__device__ void spmv_csr_scalar_dev (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, unsigned int size_vec )
{
	const unsigned int index = compute_thread_index () ;
	if ( index < size_vec ) {

		const uint2 rowptr_bounds = rowptr[index] ;
		float res = 0.0f ;

		// for each block of the block_row, mult
		for ( unsigned int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) { 
			const float xv = tex1Dfetch(texXf,colind[i]);
			res += matrix[i]*xv ;
		}
		b[index] = res ;
	}
}

__global__ void axmb_csr_krnl (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, float* r, unsigned int size_vec )
{
	const unsigned int index = compute_thread_index () ;
	if ( index < size_vec ) {

		const uint2 rowptr_bounds = rowptr[index] ;
		float res = 0.0f ;

		// for each block of the block_row, mult
		for ( unsigned int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) { 
			const float xv = tex1Dfetch(texXf,colind[i]);
			res += matrix[i]*xv ;
		}
		r[index] = b[index]  -res ;
	}
}


__device__ void axmb_csr_kernel_vector_dev (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, float* r, unsigned int size_vec )
{
#define VEC_SZ 8
	__shared__ float vals [THREAD_BLOCK_SIZE*THREAD_BLOCK_SIZE];
	const int thread_id = compute_thread_index() ; // global thread index
	const int warp_id = thread_id / VEC_SZ; // global warp index
	const int lane = thread_id & (VEC_SZ - 1); // thread index within the warp
	// one warp per row
	int row = warp_id;
	const int thread_block_id = threadIdx.x + THREAD_BLOCK_SIZE*threadIdx.y;
	const int thread_max = gridDim.x*gridDim.y*blockDim.x*blockDim.y;
	const int row_delta = thread_max/VEC_SZ;

	for(int J = 0;J<VEC_SZ;J++,row+=row_delta){
		if ( row < size_vec)
		{
			const uint2 rowptr_bounds = rowptr[row] ;
			const int row_start	=	rowptr_bounds.x;
			const int row_end	=	rowptr_bounds.y;
			// compute running sum per thread
			vals [ thread_block_id ] = 0;
			//float res = 0;
			for ( int jj = row_start + lane ; jj < row_end ; jj += VEC_SZ){
				const float xv = tex1Dfetch(texXf,colind[jj]);
				vals [ thread_block_id ] += matrix [jj] * xv;
			}

			// parallel reduction in shared memory
			//if ( lane < 16) vals [ thread_block_id ] += vals [ thread_block_id + 16];
			//if ( lane < 8) vals [ thread_block_id ] += vals [ thread_block_id + 8];
			if ( lane < 4) vals [ thread_block_id ] += vals [ thread_block_id + 4];
			if ( lane < 2) vals [ thread_block_id ] += vals [ thread_block_id + 2];
			if ( lane < 1) vals [ thread_block_id ] += vals [ thread_block_id + 1];
			// first thread writes the result
			if ( lane == 0){
				r[ row ] = b[row]- vals [ thread_block_id];
			}
		}
	}
#undef VEC_SZ
}

__global__ void axmb_csr_kernel_vector (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, float* r, unsigned int size_vec )
{
	axmb_csr_kernel_vector_dev(matrix,size_matrix,rowptr,size_rowptr,colind,size_colind,x,b,r,size_vec);
}

__device__ void axmb_csr_krnl_dev (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, float* r, unsigned int size_vec )
{
	const unsigned int index = compute_thread_index () ;
	if ( index < size_vec ) {

		const uint2 rowptr_bounds = rowptr[index] ;
		float res = 0.0f ;

		// for each block of the block_row, mult
		for ( unsigned int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) { 
			const float xv = x[colind[i]];
			res += matrix[i]*xv ;
		}
		r[index] = b[index]  -res ;
	}
}

__global__ void spmv_csr_scalar (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, unsigned int size_vec )
{
	spmv_csr_scalar_dev(matrix,size_matrix,rowptr,size_rowptr,colind,size_colind,x,b,size_vec);
}

__device__ void spmv_csr_kernel_vector_dev (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, unsigned int size_vec )
{
#define VEC_SZ 8
	__shared__ float vals [THREAD_BLOCK_SIZE*THREAD_BLOCK_SIZE];
	const int thread_id = compute_thread_index() ; // global thread index
	const int warp_id = thread_id / VEC_SZ; // global warp index
	const int lane = thread_id & (VEC_SZ - 1); // thread index within the warp
	// one warp per row
	int row = warp_id;
	const int thread_block_id = threadIdx.x + THREAD_BLOCK_SIZE*threadIdx.y;
	const int thread_max = gridDim.x*gridDim.y*blockDim.x*blockDim.y;
	const int row_delta = thread_max/VEC_SZ;

	for(int J = 0;J<VEC_SZ;J++,row+=row_delta){
		if ( row < size_vec)
		{
			const uint2 rowptr_bounds = rowptr[row] ;
			const int row_start	=	rowptr_bounds.x;
			const int row_end	=	rowptr_bounds.y;
			// compute running sum per thread
			vals [ thread_block_id ] = 0;
			//float res = 0;
			for ( int jj = row_start + lane ; jj < row_end ; jj += VEC_SZ){
				const float xv = tex1Dfetch(texXf,colind[jj]);
				vals [ thread_block_id ] += matrix [jj] * xv;
			}

			// parallel reduction in shared memory
			//if ( lane < 16) vals [ thread_block_id ] += vals [ thread_block_id + 16];
			//if ( lane < 8) vals [ thread_block_id ] += vals [ thread_block_id + 8];
			if ( lane < 4) vals [ thread_block_id ] += vals [ thread_block_id + 4];
			if ( lane < 2) vals [ thread_block_id ] += vals [ thread_block_id + 2];
			if ( lane < 1) vals [ thread_block_id ] += vals [ thread_block_id + 1];
			// first thread writes the result
			if ( lane == 0){
				b[ row ] = vals [ thread_block_id];
			}
		}
	}
#undef VEC_SZ
}

__global__ void spmv_csr_kernel_vector (
									  float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, unsigned int size_vec )
{
	spmv_csr_kernel_vector_dev(matrix,size_matrix,rowptr,size_rowptr,colind,size_colind,x,b,size_vec);

}

__global__ void memberwize_mul_kernel_float ( unsigned int size,
								   float * x,
								   float * y,
								   float * r ) {

	// Thread index
	const unsigned int index = compute_thread_index () ;
		
	if ( index < size )
		r[index] = x[index]*y[index] ;
}


/********************************************/

__device__ void memberwize_mul_device ( unsigned int size,
								   float * x,
								   float * y,
								   float * r ) {

	// Thread index
	const unsigned int index = compute_thread_index () ;
		
	if ( index < size )
		r[index] = x[index]*y[index] ;
}
//
__device__ void scopy_device ( unsigned int size,
								   float * x,
								   float * r )
{

	// Thread index
	const unsigned int index = compute_thread_index () ;
		
	if ( index < size )
		r[index] = x[index] ;
}

__device__ void saxpy_device ( unsigned int size,
								   float  a,
								   float * x,
								   float * y )
{

	// Thread index
	const unsigned int index = compute_thread_index () ;
		
	if ( index < size )
		y[index] += a*x[index] ;
}

__device__ void sscal_device ( unsigned int size,
								   float  a,
								   float * x )
{

	// Thread index
	const unsigned int index = compute_thread_index () ;
		
	if ( index < size )
		x[index] *= a ;
}
//
__device__ void swap_device(float*& x, float*& y){
	float * t = x;
	x = y;
	y = t;
}




__global__ void chebyshev_iterations_krnl(
	CRS_matrix_cuda<float> A,
	float* x, float* b,
	float* r,float *z,float*p,
	float *diag_inv,int max_iter
	)
{
	unsigned int its=0;
	float lmax = 1.f,lmin=1.f;

	float c = (lmax-lmin)/2;
	float d = (lmax+lmin)/2;
	float alpha = 0,beta = 0;
	int N = A.nrows;

	while ((int)its < max_iter)
	{
		//solve M*phat = p
		//z = linsolve(preCond,r);
		memberwize_mul_device( N, diag_inv, r, z );
		
		if(its==0){
			//Blas::copy(N,p,z);
			scopy_device(N,z,p);
			alpha = 2/d;
		}
		else{
			beta = (c*alpha/2)*(c*alpha/2);
			alpha = 1/(d-beta);
			saxpy_device(N,beta,p,z);
			//Blas::axpy(N,beta,p,z);	//z = z + beta*p;
			swap_device(p,z);	//z invalid
		}
		//x=x+alpha*p;
		saxpy_device(N,alpha,p,x);

		//Blas::spmv(N,A,x,r);
		spmv_csr_scalar_dev(
			A.a, A.nelements,
			A.rowptr,A.nrows,
			A.colind,A.nelements,
			x, r , A.nrows
		);

		// r = b - A*x
		saxpy_device(N,-1.0,b,r);
		sscal_device(N,-1.0,r);

		its++;

		//if(Blas::nrm2(N,r)<epsilon){
		//	break;
		//}	
	}
}


__global__ void chebyshev_iteration_krnl(
	CRS_matrix_cuda<float> A,
	float* x, float* b,
	float* r,float *z,float*p,
	float *diag_inv,int its
	)
{
	float lmax = 1.01f;
	float lmin=  1.f;

	float c = (lmax-lmin)/2;
	float d = (lmax+lmin)/2;
	float alpha = 0,beta = 0;
	int N = A.nrows;

		//solve M*phat = p
		memberwize_mul_device( N, diag_inv, r, z );
		
		if(its==0){
			//Blas::copy(N,p,z);
			scopy_device(N,z,p);
			alpha = 2/d;
		}
		else{
			beta = (c*alpha/2)*(c*alpha/2);
			alpha = 1/(d-beta);
			saxpy_device(N,beta,p,z);
			//Blas::axpy(N,beta,p,z);	//z = z + beta*p;
			//swap_device(p,z);	//z invalid
			scopy_device(N,z,p);
		}
		//x=x+alpha*p;
		saxpy_device(N,alpha,p,x);

}


__global__ void chebyshev_iteration_krnl_s(
	CRS_matrix_cuda<float> A,
	float* x, float* b,
	float* r,float *z,float*p,
	float *diag_inv,int cur_it,int its
	)
{
	float lmax = 1.01f;
	float lmin=  1.0f;

	float c = (lmax-lmin)/2;
	float d = (lmax+lmin)/2;
	float alpha = 0,beta = 0;
	int N = A.nrows;
	int nit = 0;
	while(nit<its){
		//solve M*phat = p
		memberwize_mul_device( N, diag_inv, r, z );
		
		if(cur_it+nit==0){
			//Blas::copy(N,p,z);
			scopy_device(N,z,p);
			alpha = 2/d;
		}
		else{
			beta = (c*alpha/2)*(c*alpha/2);
			alpha = 1/(d-beta);
			saxpy_device(N,beta,p,z);
			//Blas::axpy(N,beta,p,z);	//z = z + beta*p;
			//swap_device(p,z);	//z invalid
			scopy_device(N,z,p);
		}
		//x=x+alpha*p;
		saxpy_device(N,alpha,p,x);

		__syncthreads();


		axmb_csr_krnl_dev(
			A.a, A.nelements,
			A.rowptr,A.nrows,
			A.colind,A.nelements,
			x,b, r , A.nrows);

		//saxpy_device(N,-1.0,b,r);
		//sscal_device(N,-1.0,r);
		nit++;
	}
}

void chebyshev_iterations(CRS_matrix_cuda<float> A,float* x, float* b,
						  float* r, float*z,float* p,
						  float* diag_inv,int max_iter,
						  int thread_block_sz
						  )
{
	bind_x_texf(x,A.nrows);
	const unsigned int grid_sz= (unsigned int)(sqrt((float)A.nrows)/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,thread_block_sz);
	chebyshev_iterations_krnl<<<dim_grid,dim_block >>>(
		A,x,b,r,z,p,diag_inv,max_iter
	);
	unbind_x_tex();
}

void chebyshev_iteration(CRS_matrix_cuda<float> A,float* x, float* b,
						  float* r, float*z,float* p,
						  float* diag_inv,int its,
						  int thread_block_sz
						  )
{
	const unsigned int grid_sz= (unsigned int)(sqrt((float)A.nrows)/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,thread_block_sz);
	chebyshev_iteration_krnl<<<dim_grid,dim_block >>>(
		A,x,b,r,z,p,diag_inv,its
	);
}

void chebyshev_iteration_s(CRS_matrix_cuda<float> A,float* x, float* b,
						  float* r, float*z,float* p,
						  float* diag_inv,int cur_it, int its,
						  int thread_block_sz
						  )
{
	bind_x_texf(x,A.nrows);
	const unsigned int grid_sz= (unsigned int)(sqrt((float)A.nrows)/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,thread_block_sz);
	chebyshev_iteration_krnl_s<<<dim_grid,dim_block >>>(
		A,x,b,r,z,p,diag_inv,cur_it,its
	);
	unbind_x_tex();

}

void axmb_csr_float(CRS_matrix_cuda<float> A,
					float* x, float* b, float * r ,
					unsigned int thread_block_sz )
{
	bind_x_texf(x,A.nrows);
	const unsigned int grid_sz= (unsigned int)(sqrt((float)A.nrows)/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,thread_block_sz);
	axmb_csr_kernel_vector<<<dim_grid,dim_block >>>(
		A.a, A.nelements,
		A.rowptr,A.nrows,
		A.colind,A.nelements,
		x, b ,r, A.nrows
	);
	unbind_x_tex();
}
/***********************************************/

//Interface functions

//find b = Ax
void spmv_csr_float(CRS_matrix_cuda<float> A,
					float* x, float* b, 
					unsigned int thread_block_sz )
{
	bind_x_texf(x,A.nrows);
	const unsigned int grid_sz= (unsigned int)(sqrt((float)A.nrows)/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,thread_block_sz);
	spmv_csr_kernel_vector<<<dim_grid,dim_block >>>(
		A.a, A.nelements,
		A.rowptr,A.nrows,
		A.colind,A.nelements,
		x, b , A.nrows
	);
	unbind_x_tex();
}

void cuda_memberwise_mul_float(unsigned int N, float* x, float* y,float* z,unsigned int thread_block_sz){

	const unsigned int grid_sz= (unsigned int)(sqrt((float)N)/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,thread_block_sz);
	memberwize_mul_kernel_float<<< dim_grid,dim_block>>>(N,x,y,z);
}