//global data
texture<float, 1> texXf;

//Helper functions
__device__  unsigned int compute_thread_index () {
	return ( blockIdx.x*blockDim.x*blockDim.y+
		blockIdx.y*blockDim.x*blockDim.y*gridDim.x+
		threadIdx.x+threadIdx.y*blockDim.x) ;
}

void  inline bind_x_texf(float *x,unsigned int N)
{
	hipBindTexture(0,texXf,x,N*sizeof(float));
}

void  inline unbind_x_tex()
{
	hipUnbindTexture(texXf);
}


