#include "hip/hip_runtime.h"
//global data
texture<float, 1> texXf;

//Helper functions
__device__  unsigned int compute_thread_index () {
	return ( blockIdx.x*blockDim.x*blockDim.y+
		blockIdx.y*blockDim.x*blockDim.y*gridDim.x+
		threadIdx.x+threadIdx.y*blockDim.x) ;
}

void  inline bind_x_texf(float *x,unsigned int N)
{
	hipBindTexture(0,texXf,x,N*sizeof(float));
}

void  inline unbind_x_tex()
{
	hipUnbindTexture(texXf);
}
/*

	void spmv(const Real* x,Real* y){
		
		for (unsigned int j=0;j<dim();++j)
		{
			const Row r = operator[](j);
			Real sum = Real(0);
			for (unsigned int i=0;i<dim();++i){
				sum += r[i] * x[i];
			}
			y[j] = sum;
		}

	}*/

__global__ void spmv_dense_float(float *A, float * x, float* y, int size){
		const int i = compute_thread_index();
			if(i<size){
		
				float s = 0;
				const int abase = size*i;
				for(int j=0;j<size;++j){
					s+=A[abase + j] * x[j];
				}
				y[i] = s;
			}
}

__global__ void spmv_dense_float_tex(float *A, float * x, float* y, int size){
		const int i = compute_thread_index();
			if(i<size){
		
				float s = 0;
				const int abase = size*i;
				for(int j=0;j<size;++j){
					s+=A[abase + j] * tex1Dfetch(texXf,j);
				}
				y[i] = s;
			}
}

void spmv_dense_float(float *A, float * x, float* y, int size,unsigned int thread_block_sz)
{
	const unsigned int grid_sz= (unsigned int)(size/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,1);
	spmv_dense_float<<< dim_grid,dim_block >>>(A,x,y,size);

/*
	bind_x_texf(x,size);
	const unsigned int grid_sz= (unsigned int)(size/thread_block_sz+1); 
	dim3 dim_grid(grid_sz,grid_sz);
	dim3 dim_block(thread_block_sz,1);
	spmv_dense_float_tex<<< dim_grid,dim_block >>>(A,x,y,size);
	unbind_x_tex();*/


}

